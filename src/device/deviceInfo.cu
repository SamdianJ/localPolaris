#include "hip/hip_runtime.h"
#include "deviceInfo.cuh"

void showDeviceInfo()
{
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		std::cout << "there is no cuda device available! " << std::endl;
	}
	else
	{
		std::cout << "available device: " << deviceCount << std::endl;
	}

	int dev = 0;
	int driverVersion = 0;
	int runtimeVersion = 0;

	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Device %d:\"%s\"\n", dev, deviceProp.name);
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("  CUDA Driver Version / Runtime Version         %d.%d  /  %d.%d\n",
		driverVersion / 1000, (driverVersion % 100) / 10,
		runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	printf("  CUDA Capability Major/Minor version number:   %d.%d\n",
		deviceProp.major, deviceProp.minor);
	printf("  Total amount of global memory:                %.2f GBytes (%llu bytes)\n",
		(float)deviceProp.totalGlobalMem / pow(1024.0, 3), deviceProp.totalGlobalMem);
	printf("  GPU Clock rate:                               %.0f MHz (%0.2f GHz)\n",
		deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
	printf("  Memory Bus width:                             %d-bits\n",
		deviceProp.memoryBusWidth);
	if (deviceProp.l2CacheSize)
	{
		printf("  L2 Cache Size:                            	%d bytes,  %d KB\n",
			deviceProp.l2CacheSize, deviceProp.l2CacheSize / 1024);
	}
	printf("  Max Texture Dimension Size (x,y,z)            1D=(%d),2D=(%d,%d),3D=(%d,%d,%d)\n",
		deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1]
		, deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
	printf("  Max Layered Texture Size (dim) x layers       1D=(%d) x %d,2D=(%d,%d) x %d\n",
		deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
		deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
		deviceProp.maxTexture2DLayered[2]);
	printf("  Total amount of constant memory               %lu bytes,  %lu KB\n",
		deviceProp.totalConstMem, deviceProp.totalConstMem / 1024);
	printf("  Total amount of shared memory per block:      %lu bytes,  %lu KB\n",
		deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerBlock / 1024);
	printf("  Total number of registers available per block:%d bytes,  %lu KB\n",
		deviceProp.regsPerBlock, deviceProp.regsPerBlock / 1024);
	printf("  Wrap size:                                    %d\n", deviceProp.warpSize);
	printf("  Maximun number of thread per multiprocesser:  %d\n",
		deviceProp.maxThreadsPerMultiProcessor);
	printf("  Maximun number of thread per block:           %d\n",
		deviceProp.maxThreadsPerBlock);
	printf("  Maximun size of each dimension of a block:    %d x %d x %d\n",
		deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("  Maximun size of each dimension of a grid:     %d x %d x %d\n",
		deviceProp.maxGridSize[0],
		deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	printf("  Maximu memory pitch                           %lu bytes\n", deviceProp.memPitch);
	printf("----------------------------------------------------------\n");
	printf("Number of multiprocessors:                      %d\n", deviceProp.multiProcessorCount);
	printf("Total amount of constant memory:                %4.2f KB\n",
		deviceProp.totalConstMem / 1024.0);
	printf("Total amount of shared memory per block:        %4.2f KB\n",
		deviceProp.sharedMemPerBlock / 1024.0);
	printf("Total number of registers available per block:  %d\n",
		deviceProp.regsPerBlock);
	printf("Warp size                                       %d\n", deviceProp.warpSize);
	printf("Maximum number of threads per block:           %d\n",
		deviceProp.maxThreadsPerBlock);
	printf("Maximum number of threads per multiprocessor:  %d\n",
		deviceProp.maxThreadsPerMultiProcessor);
	printf("Maximum number of warps per multiprocessor:     %d\n",
		deviceProp.maxThreadsPerMultiProcessor / 32);

}